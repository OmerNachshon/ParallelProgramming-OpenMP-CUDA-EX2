#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>





__global__ void heavyKernel(double* arr, int size, int loopSize, double* answer) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        double sum = heavy(arr[i], loopSize);
        atomicAdd(answer, sum);
    }

}


double heavy(double data, int loopSize) {
    double sum = 0;       
    for (int i = 0; i < loopSize; i++)
        sum += cos(exp(sin(data * (i % 11))))/ loopSize;

     return sum;             
}


int main(int argc, char *argv[]) {
    int size = atoi(argv[1]);
    int loopSize = atoi(argv[2]);
    double* arr = (double*)malloc(size * sizeof(double));
    double* dev_arr;
    double* dev_answer;
    double answer = 0;
    hipEvent_t start, stop;

    for (int i = 0; i < size; i++)
        arr[i] = rand() / RAND_MAX;

    hipMalloc(&dev_arr, size * sizeof(double));
    hipMalloc(&dev_answer, sizeof(double));
    hipMemcpy(dev_arr, arr, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(dev_answer, 0, sizeof(double));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    heavyKernel<<<(size + 255) / 256, 256>>>(dev_arr, size, loopSize, dev_answer);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipMemcpy(&answer, dev_answer, sizeof(double), hipMemcpyDeviceToHost);
    printf("answer = %e\n", answer);
    
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time taken: %f ms\n", elapsedTime);

    hipFree(dev_arr);
    hipFree(dev_answer);
    free(arr);

    return 0;
}

 