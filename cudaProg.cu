#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__device__ double heavy(double data, int loopSize) {
    double sum = 0;       
    for (int i = 0; i < loopSize; i++)
        sum += cos(exp(sin(data * (i % 11))))/ loopSize;

     return sum;             
}


__global__ void heavyKernel(double* arr, int size, int loopSize, double* sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        sum[i] = heavy(arr[i], loopSize);
        
    }

}

int main(int argc, char *argv[]) {
    int size = atoi(argv[1]);
    int loopSize = atoi(argv[2]);
    double* arr = (double*)malloc(size * sizeof(double));
    double* answers = (double*)malloc(size * sizeof(double));
    double* dev_arr;
    double* dev_answer;
    double sum = 0;
    hipEvent_t start, stop;

//random value for arr 
    for (int i = 0; i < size; i++)
        arr[i] = rand() / RAND_MAX;

    hipMalloc(&dev_arr, size * sizeof(double));
    hipMalloc(&dev_answer, sizeof(double));
    hipMemcpy(dev_arr, arr, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_answer, answers, sizeof(double), hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    heavyKernel<<<(size + 255) / 256, 256>>>(dev_arr, size, loopSize, dev_answer);
    hipMemcpy(answers, dev_answer, sizeof(double), hipMemcpyDeviceToHost);

//sum up all values into variable sum 
    for(int i=0;i<size;i++)
    sum+=answers[i];

    printf("answer = %e\n", sum);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    //print total runtime
    printf("Time taken: %f s\n", elapsedTime/1000);

    hipFree(dev_arr);
    hipFree(dev_answer);
    free(arr);

    return 0;
}

 

 